#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hipfft/hipfft.h>

#define GROUP 1
#include <chrono>

int main()
{
    const int num_benchmark_samples_2D = 9;
    const int num_benchmark_samples_3D = 11;
    const int num_runs = 5;
    //cuFFT works best in when last dimension is the longest in R2C mode
    int benchmark_dimensions_2D[num_benchmark_samples_2D][4] = { {32, 32, 1, 2}, {64, 64, 1, 2}, {32, 256, 1, 2}, {256, 256, 1, 2}, {256, 1024, 1, 2},{1024, 1024, 1, 2}, {256, 4096, 1, 2}, {2048, 4096, 1, 2}, {4096, 4096, 1, 2} };
    int benchmark_dimensions_3D[num_benchmark_samples_3D][4] = { {32, 32, 32, 3}, {64, 64, 64, 3}, {32, 32, 256, 3}, {32, 256, 256, 3}, {256, 256, 256, 3}, {32, 256, 1024, 3}, {8, 1024, 1024, 3}, {8, 1024, 2048, 3}, {256, 256, 2048, 3}, {8, 4096, 4096, 3}, {32, 4096, 4096, 3} };

    for (int n = 0; n < num_benchmark_samples_2D; n++) {

        for (int r = 0; r < num_runs; r++) {
            hipfftHandle planR2C;
            hipfftHandle planC2R;
            hipfftComplex* dataC;
            hipfftReal* dataR;

            hipfftReal* inputReal;
            int dims[2] = { benchmark_dimensions_2D[n][0] , benchmark_dimensions_2D[n][1] };

            inputReal = (hipfftReal*)(malloc(sizeof(hipfftReal) * dims[0] * dims[1]));
            
            for (int j = 0; j < dims[1]; j++) {
                for (int i = 0; i < dims[0]; i++) {
                    inputReal[i+j* dims[0]] = j;
                }
            }
            hipMalloc((void**)&dataC, sizeof(hipfftComplex) * dims[0] * (dims[1] / 2 + 1));
            hipMalloc((void**)&dataR, sizeof(hipfftReal) * dims[0] * dims[1]);
            hipMemcpy(dataR, inputReal, sizeof(hipfftReal) * dims[0] * dims[0], hipMemcpyHostToDevice);
            if (hipGetLastError() != hipSuccess) {
                fprintf(stderr, "Cuda error: Failed to allocate\n");
                return;
            }

            if (hipfftPlanMany(&planC2R, 2, dims,
                NULL, 1, 0,
                NULL, 1, 0,
                HIPFFT_C2R, GROUP) != HIPFFT_SUCCESS) {
                fprintf(stderr, "CUFFT Error: Unable to create C2R plan\n");
                return;
            }
            if (hipfftPlanMany(&planR2C, 2, dims,
                NULL, 1, 0,
                NULL, 1, 0,
                HIPFFT_R2C, GROUP) != HIPFFT_SUCCESS) {
                fprintf(stderr, "CUFFT Error: Unable to create R2C plan\n");
                return;
            }
            
            double totTime = 0;
            int batch = ((512.0 * 1024.0 * 1024.0) / dims[0]/(dims[1]/2+1) > 1000) ? 1000 : (512.0 * 1024.0 * 1024.0) / dims[0] / (dims[1]/2+1);
            if (batch == 0) batch = 1;

            auto timeSubmit = std::chrono::steady_clock::now();
            for (int i = 0; i < batch; i++) {

                hipfftExecR2C(planR2C, dataR, dataC);
                hipDeviceSynchronize();
                hipfftExecC2R(planC2R, dataC, dataR);
                hipDeviceSynchronize();
               
            }
            auto timeEnd = std::chrono::steady_clock::now();
            totTime = (std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSubmit).count() * 0.001)/batch;

            printf("System: %dx%dx%d, run: %d, Buffer: %d MB, time per step: %0.3f ms, batch: %d\n", dims[1], dims[0], 1, r, (sizeof(hipfftReal) * dims[0] * dims[1]+ sizeof(hipfftComplex) * dims[0] * (dims[1] / 2 + 1)) / 1024 / 1024, totTime, batch);
               
            hipfftDestroy(planR2C);
            hipFree(dataR);
            hipfftDestroy(planC2R);
            hipFree(dataC);
        }
    }
    for (int n = 0; n < num_benchmark_samples_3D; n++) {

        for (int r = 0; r < num_runs; r++) {
            hipfftHandle planR2C;
            hipfftHandle planC2R;
            hipfftComplex* dataC;
            hipfftReal* dataR;

            hipfftReal* inputReal;
            int dims[3] = { benchmark_dimensions_3D[n][0] , benchmark_dimensions_3D[n][1] , benchmark_dimensions_3D[n][2] };

            inputReal = (hipfftReal*)(malloc(sizeof(hipfftReal) * dims[0] * dims[1] * dims[2]));
            for (int k = 0; k < dims[2]; k++) {
                for (int j = 0; j < dims[1]; j++) {
                    for (int i = 0; i < dims[0]; i++) {
                        inputReal[i + j * dims[0]+k*dims[0]*dims[1]] = k;
                    }
                }
            }
            hipMalloc((void**)&dataC, sizeof(hipfftComplex) * dims[0] * dims[1] * (dims[2] / 2 + 1));
            hipMalloc((void**)&dataR, sizeof(hipfftReal) * dims[0] * dims[1] * dims[2]);
            hipMemcpy(dataR, inputReal, sizeof(hipfftReal) * dims[0] * dims[0] * dims[2], hipMemcpyHostToDevice);
            if (hipGetLastError() != hipSuccess) {
                fprintf(stderr, "Cuda error: Failed to allocate\n");
                return;
            }

            if (hipfftPlanMany(&planC2R, 3, dims,
                NULL, 1, 0,
                NULL, 1, 0,
                HIPFFT_C2R, GROUP) != HIPFFT_SUCCESS) {
                fprintf(stderr, "CUFFT Error: Unable to create C2R plan\n");
                return;
            }
            if (hipfftPlanMany(&planR2C, 3, dims,
                NULL, 1, 0,
                NULL, 1, 0,
                HIPFFT_R2C, GROUP) != HIPFFT_SUCCESS) {
                fprintf(stderr, "CUFFT Error: Unable to create R2C plan\n");
                return;
            }

            double totTime = 0;
            int batch = ((512.0 * 1024.0 * 1024.0) / dims[0] / dims[1] / (dims[2]/2+1)> 1000) ? 1000 : (512.0 * 1024.0 * 1024.0) / dims[0] / dims[1] / (dims[2]/2+1);
            if (batch == 0) batch = 1;

            auto timeSubmit = std::chrono::steady_clock::now();
            for (int i = 0; i < batch; i++) {

                hipfftExecR2C(planR2C, dataR, dataC);
                hipDeviceSynchronize();
                hipfftExecC2R(planC2R, dataC, dataR);
                hipDeviceSynchronize();

            }
            auto timeEnd = std::chrono::steady_clock::now();
            totTime = (std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSubmit).count() * 0.001) / batch;

            printf("System: %dx%dx%d, run: %d, Buffer: %d MB, time per step: %0.3f ms, batch: %d\n", dims[2], dims[1], dims[0], r, (sizeof(hipfftReal) * dims[0] * dims[1] * dims[2]+ sizeof(hipfftComplex) * dims[0] * dims[1] * (dims[2] / 2 + 1)) / 1024 / 1024, totTime, batch);

            hipfftDestroy(planR2C);
            hipFree(dataR);
            hipfftDestroy(planC2R);
            hipFree(dataC);
        }
    }
}
